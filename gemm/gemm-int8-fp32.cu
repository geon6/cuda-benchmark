// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <argparse.hpp>
#include <chrono>
#include <iostream>

// 3072 6400 7168
int M = 3072;
int N = 6400;
int K = 7168;

int device_id = 0;

// A40的性能
double int8_gflops = 299.3 * 1024.0;
double int8_tflops = 299.3;

// 记录最好的算法, layout
double best_tflops = 0.0;
int best_algo, best_layout_a, best_layout_b;

struct PrecisionConfig {
  std::string func_name;
  hipDataType cudaType;
  hipblasComputeType_t cublasType;
  int bytesPerElement;
  std::string type_name;
  int NUM_ITERATIONS;
  int WARMUP_ITERATIONS = 10;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  hipblasGemmAlgo_t algo;
};

void test(const PrecisionConfig& config) {
  hipSetDevice(device_id);
  int8_t *d_A, *d_B;
  float* d_C;
  hipMallocManaged(&d_A, M * K * config.bytesPerElement);
  hipMallocManaged(&d_B, K * N * config.bytesPerElement);
  hipMallocManaged(&d_C, M * N * sizeof(float));

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int alpha = 1;
  int beta = 0;

  for (int i = 0; i < config.WARMUP_ITERATIONS; ++i) {
    hipblasGemmEx(handle, config.transa, config.transb, M, N, K, &alpha, d_A,
                 config.cudaType, (config.transa == HIPBLAS_OP_N ? M : K), d_B,
                 config.cudaType, (config.transb == HIPBLAS_OP_N ? K : N), &beta,
                 d_C, HIP_R_32F, M, config.cublasType, config.algo);
  }

  hipError_t syncError = hipDeviceSynchronize();
  auto start = std::chrono::high_resolution_clock::now();

  if (syncError != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
  }

  for (int i = 0; i < config.NUM_ITERATIONS; ++i) {
    hipblasGemmEx(handle, config.transa, config.transb, M, N, K, &alpha, d_A,
                 config.cudaType, (config.transa == HIPBLAS_OP_N ? M : K), d_B,
                 config.cudaType, (config.transb == HIPBLAS_OP_N ? K : N), &beta,
                 d_C, HIP_R_32F, M, config.cublasType, config.algo);
  }
  syncError = hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();

  if (syncError != hipSuccess) {
    std::cout << "CUDA error: " << hipGetErrorString(syncError) << std::endl;
  }
  auto dt = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  // std::cout << "Average " << config.name << " Single Op Duration: "
  // << dt.count() / config.NUM_ITERATIONS << " us" << std::endl;

  double time_second = dt.count() / 1.0e6;
  double ops = 2.0 * M * N * K * config.NUM_ITERATIONS;
  double OPS = ops / time_second;
  double TOPS = OPS / 1.0e12;
  double ratio = TOPS / int8_tflops;

  if (TOPS > best_tflops) {
    best_tflops = TOPS;
    best_algo = (int)config.algo;
    best_layout_a = (int)config.transa;
    best_layout_b = (int)config.transb;
  }

  std::cout << config.func_name << "," << config.type_name << "," << config.algo
            << "," << config.transa << "," << config.transb << "," << TOPS
            << "," << ratio << std::endl;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipblasDestroy(handle);
}

int main(int argc, char** argv) {
  argparse::ArgumentParser program("gemm-int8-fp32");

  // append模式不会输出csv第一行
  program.add_argument("--append")
      .flag()
      .help("output the first line of csv file?");
  program.add_argument("-m").default_value(512).store_into(M).help("set m");
  program.add_argument("-k").default_value(512).store_into(K).help("set k");
  program.add_argument("-n").default_value(512).store_into(N).help("set n");
  program.add_argument("--device_id")
      .default_value(0)
      .store_into(device_id)
      .help("set the device id");
  program.parse_args(argc, argv);

  PrecisionConfig int8 = {
      .func_name = "hipblasGemmEx",
      .cudaType = HIP_R_8I,
      .cublasType = HIPBLAS_COMPUTE_32F,
      .bytesPerElement = sizeof(int8_t),
      .type_name = "int8",
      .NUM_ITERATIONS = 100,
      .WARMUP_ITERATIONS = 10,
      // .transa = HIPBLAS_OP_N,
      // .transb = HIPBLAS_OP_T,
      // .algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP
  };

  if (program["--append"] == false)
    std::cout << "func_name,dtype,algo,layout_a,layout_b,TOPS,ratio"
              << std::endl;

  for (int layout_a = 0; layout_a <= 1; layout_a++) {
    int8.transa = (hipblasOperation_t)layout_a;
    for (int layout_b = 0; layout_b <= 1; layout_b++) {
      int8.transb = (hipblasOperation_t)layout_b;
      for (int i = -1; i <= 23; i++) {
        int8.algo = (hipblasGemmAlgo_t)i;
        test(int8);
      }

      for (int i = 99; i <= 115; i++) {
        int8.algo = (hipblasGemmAlgo_t)i;
        test(int8);
      }
    }
  }

  std::cout << "best tflops: " << best_tflops << "("
            << best_tflops / int8_tflops << ")" << ", best algo: " << best_algo
            << ", best layout a: " << best_layout_a
            << ", best layout b: " << best_layout_b << std::endl;

  return 0;
}